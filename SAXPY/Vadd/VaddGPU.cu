#include "hip/hip_runtime.h"
#include "GPUErrors.h"
#include "Vadd.h"

__global__ void VectAdd(float* g_A, float* g_B, float* g_C, int size){
    int idx=threadIdx.x+(blockDim.x*blockIdx.x);
    if(idx<size){
        g_C[idx]=g_A[idx]+g_B[idx];
    }
}

__host__ void gpuVaddHelper(float* h_A, float* h_B, float* h_C, float* ref, const int size){
    float* d_A, * d_B, * d_C;
	const int SizeInBytes = size* sizeof(float);
	float ElapsedTime{};
	float ElapsedTime_dh{};
	float ElapsedTime_hd{};
    hipEvent_t start, stop;
    hipEvent_t start_dh, stop_dh;
    hipEvent_t start_hd, stop_hd;
    int t_B=256;
    int b_G=ceil(1.0*size/t_B);
    HandleCUDAError(hipEventCreate(&start));
    HandleCUDAError(hipEventCreate(&stop));
    HandleCUDAError(hipEventCreate(&start_dh));
    HandleCUDAError(hipEventCreate(&stop_dh));
    HandleCUDAError(hipEventCreate(&start_hd));
    HandleCUDAError(hipEventCreate(&stop_hd));
    HandleCUDAError(hipMalloc((void**)&d_A,SizeInBytes));
    HandleCUDAError(hipMalloc((void**)&d_B,SizeInBytes));
    HandleCUDAError(hipMalloc((void**)&d_C,SizeInBytes));

    HandleCUDAError(hipEventRecord(start_hd,0));
    HandleCUDAError(hipMemcpy(d_A,h_A,SizeInBytes,hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(d_B,h_B,SizeInBytes,hipMemcpyHostToDevice));
    if (!HandleCUDAError(hipEventRecord(stop_hd, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop_hd))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&ElapsedTime_hd, start_hd, stop_hd))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    HandleCUDAError(hipEventRecord(start,0));
    VectAdd<<<b_G,t_B>>>(d_A,d_B,d_C,size);
    hipDeviceSynchronize();
    if (!HandleCUDAError(hipEventRecord(stop, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&ElapsedTime, start, stop))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    cout<<"GPU Vadd: "<<ElapsedTime<<" ms"<<endl;
    HandleCUDAError(hipEventRecord(start_dh,0));
    HandleCUDAError(hipMemcpy(h_C,d_C,SizeInBytes,hipMemcpyDeviceToHost));
    if (!HandleCUDAError(hipEventRecord(stop_dh, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop_dh))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&ElapsedTime_dh, start_dh, stop_dh))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    cout<<"Memcpy time: "<<ElapsedTime_dh+ElapsedTime_hd<<" ms"<<endl;
    VaddVerification(ref,h_C,size);
    HandleCUDAError(hipEventDestroy(start));
	HandleCUDAError(hipEventDestroy(stop));
	HandleCUDAError(hipEventDestroy(start_dh));
	HandleCUDAError(hipEventDestroy(stop_dh));
	HandleCUDAError(hipEventDestroy(start_hd));
	HandleCUDAError(hipEventDestroy(stop_hd));
    HandleCUDAError(hipFree(d_A));
    HandleCUDAError(hipFree(d_B));
    HandleCUDAError(hipFree(d_C));
    HandleCUDAError(hipDeviceReset());
}