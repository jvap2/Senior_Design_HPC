#include "hip/hip_runtime.h"
#include "GPUErrors.h"
#include "Vadd.h"

__global__ void VectAdd(float* g_A, float* g_B, float* g_C, int size){
    int idx=threadIdx.x+(blockDim.x*blockIdx.x);
    if(idx<size){
        g_C[idx]=g_A[idx]+g_B[idx];
    }
}

__host__ void gpuVaddHelper(float* h_A, float* h_B, float* h_C, float* ref, const int size){
    float* d_A, * d_B, * d_C;
	const int SizeInBytes = size* sizeof(float);
	float ElapsedTime{};
    hipEvent_t start, stop;
    int t_B=256;
    int b_G=ceil(1.0*size/t_B);
    HandleCUDAError(hipEventCreate(&start));
    HandleCUDAError(hipEventCreate(&stop));
    HandleCUDAError(hipMalloc((void**)&d_A,SizeInBytes));
    HandleCUDAError(hipMalloc((void**)&d_B,SizeInBytes));
    HandleCUDAError(hipMalloc((void**)&d_C,SizeInBytes));

    HandleCUDAError(hipEventRecord(start,0));
    HandleCUDAError(hipMemcpy(d_A,h_A,SizeInBytes,hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(d_B,h_B,SizeInBytes,hipMemcpyHostToDevice));
    if (!HandleCUDAError(hipEventRecord(stop, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&ElapsedTime, start, stop))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    cout<<"Memcpy from H=>D: "<<ElapsedTime<<" ms"<<endl;
    HandleCUDAError(hipEventRecord(start,0));
    VectAdd<<<b_G,t_B>>>(d_A,d_B,d_C,size);
    hipDeviceSynchronize();
    if (!HandleCUDAError(hipEventRecord(stop, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&ElapsedTime, start, stop))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    cout<<"GPU Vadd: "<<ElapsedTime<<" ms"<<endl;
    HandleCUDAError(hipEventRecord(start,0));
    HandleCUDAError(hipMemcpy(h_C,d_C,SizeInBytes,hipMemcpyDeviceToHost));
    if (!HandleCUDAError(hipEventRecord(stop, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&ElapsedTime, start, stop))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    cout<<"Memcpy from D=>H: "<<ElapsedTime<<" ms"<<endl;
    VaddVerification(ref,h_C,size);
    HandleCUDAError(hipFree(d_A));
    HandleCUDAError(hipFree(d_B));
    HandleCUDAError(hipFree(d_C));
    HandleCUDAError(hipDeviceReset());
}