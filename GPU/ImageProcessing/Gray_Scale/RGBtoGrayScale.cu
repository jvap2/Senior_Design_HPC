#include "hip/hip_runtime.h"
#include "RGBtoGrayScale.h"
//Kernel Version 0
__global__ void gpu_RGBtoGrayScaleVer0(unsigned char* in, unsigned char* out, unsigned int h, unsigned int w)
{
	unsigned int idx = threadIdx.x + (blockDim.x * blockIdx.x);
	if (idx < (h * w)) {
		out[idx] = .21f * in[idx] + .71f * in[idx + (h * w)] + .07 * in[idx + 2 * h * w];
	}
}
//Kernel Version 1
__global__ void gpu_RGBtoGrayScaleVer1(unsigned char* r,
	unsigned char* g,
	unsigned char* b,
	unsigned char* out, unsigned int h, unsigned int w)
{
	unsigned int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	if (idx < (h * w)) {
		out[idx] = 0.21f * *(r + idx) + 0.72f * *(g + idx) + .07f * *(b + idx);//This allows us to avoid multiplying 2 times offset
	}
}

//Kernel Version Two; 2D Implementation
__global__ void gpu_RGBtoGrayScaleVer2(unsigned char* r,
	unsigned char* g,
	unsigned char* b,
	unsigned char* out, unsigned int h, unsigned int w) {
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	//Compute the gray scale image index
	int idx = y * w+x;//y*w is row times width, so this points to the beginning of each row, x is the offset to get to a column
	if (x<w && y<h) {
		out[idx] = 0.21f * *(r + idx) + 0.72f * *(g + idx) + .07f * *(b + idx);//We can use this since we converted 2d to 1d
	}
	//When we are closer to the hardware we need to write this in 1D

}

//Host Helper function
__host__ void gpu_RGBtoGrayScaleHelper(unsigned char* h_in, unsigned char* h_out, 
	unsigned int rgbSIZE,
	unsigned int graySIZE,
	unsigned int h,
	unsigned int w,
	unsigned int kernelVer)
{
	float ElapsedTime{}, Elapsed_hd{}, Elapsed_dh{};
	double computeTime;
	unsigned char* d_in, * d_out;
	//Allocating device memory for the RGB and GrayScale Images
	if (!HandleCUDAError(hipMalloc((void**) &d_in,rgbSIZE))) {
		cout << "Error allocating on the GPU for the RGB image" << endl;
	}
	//Allocate Memory on the GPU for Gray Scale
	if (!HandleCUDAError(hipMalloc((void**)&d_out, graySIZE))) {
		cout << "Error allocating on the GPU for the gray scale image" << endl;
	}
	hipEvent_t start_ker, stop_ker;
    hipEvent_t start_dh, stop_dh;
    hipEvent_t start_hd, stop_hd;
	HandleCUDAError(hipEventCreate(&start_ker));
	HandleCUDAError(hipEventCreate(&stop_ker));
    HandleCUDAError(hipEventCreate(&start_dh));
    HandleCUDAError(hipEventCreate(&stop_dh));
    HandleCUDAError(hipEventCreate(&start_hd));
    HandleCUDAError(hipEventCreate(&stop_hd));
    HandleCUDAError(hipEventRecord(start_hd,0));
	//Copying the RGB image to the device
	if (!HandleCUDAError(hipMemcpy(d_in,h_in,rgbSIZE,hipMemcpyHostToDevice))) {
		cout << "Error transferring RGB image from host to device" << endl;
	}
	if (!HandleCUDAError(hipEventRecord(stop_hd, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop_hd))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&Elapsed_hd, start_hd, stop_hd))) {
		cout << "Unable to find elapsed time between events" << endl;
	}

	//Setup Execution Configuration Parameters
	unsigned int threadsPerBlock = 256;//We are choosing 256 threads for 8 warps
	unsigned int blocksPerGrid = ((w * h) / threadsPerBlock)+1;//We are adding one since the the division is rounding down, and we need to account for extras
	
	cout << "Image Grid Size = " << (w * h) << " pixels" << endl;
	cout << "Number of threads per block = " << threadsPerBlock << endl;
	cout << "Number of blocks per Grid = " << blocksPerGrid << endl;
	cout << "Total Number of Threads in the Grid = " << threadsPerBlock * blocksPerGrid << endl;
	
	if (kernelVer == 0)
	{
		//Launch the RGB to Gray Scale Kernel - Ver 0
		gpu_RGBtoGrayScaleVer0 << <blocksPerGrid, threadsPerBlock >> > (d_in,
			d_out,
			h,
			w);
		hipDeviceSynchronize();

	}
	else if (kernelVer==1)
	{
		unsigned char* d_r = d_in;//We are pointing to the red
		unsigned char* d_g = d_in+(h*w);
		unsigned char* d_b = d_in + (2*h * w);
		//Launch the RGB to Gray Scale Kernel - Ver 1
		auto start = high_resolution_clock::now();//Comes from std::chrono namespace
		HandleCUDAError(hipEventRecord(start_ker,0));
		gpu_RGBtoGrayScaleVer1<< <blocksPerGrid, threadsPerBlock >> > (d_r,
			d_g,
			d_b,
			d_out,
			h,
			w);
		hipDeviceSynchronize();
		if (!HandleCUDAError(hipEventRecord(stop_ker, 0))) {
			cout << "Unable to perform event records for stop" << endl;
		}
		//Synchronize the stop event
		if (!HandleCUDAError(hipEventSynchronize(stop_ker))) {
			cout << "Unable to perform stream synch with stream_4" << endl;
		}
		//Save the elapsed time
		if (!HandleCUDAError(hipEventElapsedTime(&ElapsedTime, start_ker, stop_ker))) {
			cout << "Unable to find elapsed time between events" << endl;
		}

		
	}
	else {
		unsigned char* d_r = d_in;//We are pointing to the red
		unsigned char* d_g = d_in + (h * w);
		unsigned char* d_b = d_in + (2 * h * w);
		//Setup Execution Configuration Parameters
		int TILE_WIDTH = 16;
		dim3 dimGrid(ceil((float)w / TILE_WIDTH), ceil((float)h / TILE_WIDTH));
		dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);//Each block has 16x16 threads
		auto start = high_resolution_clock::now();
		gpu_RGBtoGrayScaleVer2 << <dimGrid, dimBlock >> > (d_r,
			d_g,
			d_b,
			d_out,
			h,
			w);
		hipDeviceSynchronize();
		auto end = high_resolution_clock::now();
		auto elasped_seconds = end - start;
		computeTime = duration_cast<microseconds>(elasped_seconds).count();
	}
	//Copy the grayscale image data from device to host
	HandleCUDAError(hipEventRecord(start_dh,0));
	if (!HandleCUDAError(hipMemcpy(h_out, d_out, graySIZE, hipMemcpyDeviceToHost))) {
		cout << "Error transferring RGB image from host to device" << endl;
	}
	if (!HandleCUDAError(hipEventRecord(stop_dh, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop_dh))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&Elapsed_dh, start_dh, stop_dh))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
	float total_time=Elapsed_dh+Elapsed_hd;
    float bytes_transferred=(rgbSIZE+graySIZE)*1.0f;
    float throughput=(bytes_transferred*1e-6)/(total_time);
    cout<< "GPU CG Memory elapsed time: "<<total_time<< " ms"<<endl;
    cout<< "GPU CG Exec elapsed time: "<<ElapsedTime<< " ms"<<endl;
    cout<< "GPU CG total elapsed time: "<<ElapsedTime+total_time<< " ms"<<endl;
    cout<<"GPU throughput: "<<throughput<< "GB/s"<<endl;
	if (!HandleCUDAError(hipFree(d_in)))
	{
		cout << "Error freeing RGB image memory" << endl;
	}
	if (!HandleCUDAError(hipFree(d_out)))
	{
		cout << "Error freeing GrayScale image memory" << endl;
	}
	HandleCUDAError(hipDeviceReset());
}