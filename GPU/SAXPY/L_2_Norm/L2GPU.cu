#include "hip/hip_runtime.h"
#include "GPUErrors.h"
#include "L2.h"


__global__ void d_L_2_Partial_Reduction(float* in, float* hold_vect,float* g_PartialSum, int size){
    int idx=threadIdx.x+(blockDim.x*blockIdx.x);
    int tid=threadIdx.x;
    if (idx>=size){
        return;
    }
    hold_vect[idx]=in[idx]*in[idx];
    __syncthreads();
    float* blockAddress=hold_vect+(blockDim.x*blockIdx.x);
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
	{
		if (tid < stride)
		{
			//tid<stride ensures we do not try to access memory past the vector allocated to the block
			//tid+stride<size allows for vector sizes less than blockDim
			blockAddress[tid] += blockAddress[tid + stride];
		}
		__syncthreads();//Make all of the threads wait to go to the next iteration so the values are up to date
	}
    if(tid==0){
        g_PartialSum[blockIdx.x]=blockAddress[0];
    }
}

__global__ void d_Commit_L_2(float* g_Partial_Sum, float* mu){
    int tid=threadIdx.x;
    float* blockAddress=g_Partial_Sum;
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
	{
		if (tid < stride)
		{
			//tid<stride ensures we do not try to access memory past the vector allocated to the block
			//tid+stride<size allows for vector sizes less than blockDim
			blockAddress[tid] += blockAddress[tid + stride];
		}
		__syncthreads();//Make all of the threads wait to go to the next iteration so the values are up to date
	}
    if(tid==0){
        *(mu)=sqrtf(blockAddress[0]);
    }
}


__host__ void L_2_Helper(float* vector, float* ref,float* mu_GPU, int size){
    float ElapsedTime{}, Elapsed_hd{}, Elapsed_dh{};
    float* d_v;
    float* d_p_sum;
    float* d_hold_sum;
    float* d_mu;
    int vec_size=size*sizeof(float);
    int threads_p_block=256;
    int blocks_per_grid=size/threads_p_block+1;
    if(blocks_per_grid>1024){
        return;
    }
    hipEvent_t start, stop;
    hipEvent_t start_dh, stop_dh;
    hipEvent_t start_hd, stop_hd;
    HandleCUDAError(hipEventCreate(&start));
	HandleCUDAError(hipEventCreate(&stop));
    HandleCUDAError(hipEventCreate(&start_dh));
    HandleCUDAError(hipEventCreate(&stop_dh));
    HandleCUDAError(hipEventCreate(&start_hd));
    HandleCUDAError(hipEventCreate(&stop_hd));
    int P_sum_size=blocks_per_grid*sizeof(float);
    HandleCUDAError(hipMalloc((void**)&d_v,vec_size));
    HandleCUDAError(hipMalloc((void**)&d_p_sum,P_sum_size));
    HandleCUDAError(hipMalloc((void**)&d_hold_sum, vec_size));
    HandleCUDAError(hipMalloc((void**)&d_mu,sizeof(float)));
    HandleCUDAError(hipEventRecord(start_hd,0));
    HandleCUDAError(hipMemcpy(d_v,vector,vec_size,hipMemcpyHostToDevice));
    if (!HandleCUDAError(hipEventRecord(stop_hd, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop_hd))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&Elapsed_hd, start_hd, stop_hd))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    HandleCUDAError(hipEventRecord(start,0));
    d_L_2_Partial_Reduction<<<blocks_per_grid,threads_p_block>>>(d_v,d_hold_sum,d_p_sum,size);
    hipDeviceSynchronize();
    d_Commit_L_2<<<1,blocks_per_grid>>>(d_p_sum,d_mu);
    hipDeviceSynchronize();
    if (!HandleCUDAError(hipEventRecord(stop, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&ElapsedTime, start, stop))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    HandleCUDAError(hipEventRecord(start_dh,0));
    HandleCUDAError(hipMemcpy(mu_GPU,d_mu,sizeof(float),hipMemcpyDeviceToHost));
    if (!HandleCUDAError(hipEventRecord(stop_dh, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop_dh))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&Elapsed_dh, start_dh, stop_dh))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    float total_time=Elapsed_dh+Elapsed_hd;
    float bytes_transferred=(vec_size+sizeof(float))*1.0f;
    float throughput=(bytes_transferred*1e-6)/(total_time);
    cout<<"Device to Host: "<<Elapsed_dh<<" ms"<<endl;
    cout<<"Host to Device: "<<Elapsed_hd<<" ms"<<endl;
    cout<< "GPU CG Memory elapsed time: "<<total_time<< " ms"<<endl;
    cout<< "GPU CG Exec elapsed time: "<<ElapsedTime<< " ms"<<endl;
    cout<< "GPU CG total elapsed time: "<<ElapsedTime+total_time<< " ms"<<endl;
    cout<<"GPU throughput: "<<throughput<< "GB/s"<<endl;
    Verify(mu_GPU,ref);
    HandleCUDAError(hipFree(d_v));
    HandleCUDAError(hipFree(d_p_sum));
    HandleCUDAError(hipFree(d_hold_sum));
    HandleCUDAError(hipFree(d_mu));
    hipDeviceReset();
}