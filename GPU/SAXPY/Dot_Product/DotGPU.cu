#include "hip/hip_runtime.h"
#include "GPUErrors.h"
#include "Dot.h"


__global__ void d_Dot_Partial(float* in_1, float* in_2, float* hold, float* g_PartialSums, int size){
    int idx=threadIdx.x+(blockDim.x*blockIdx.x);
    int tid=threadIdx.x;
    hold[idx]=in_1[idx]*in_2[idx];
    __syncthreads();
    if (idx>=size){
        return;
    }
    float* blockAddress= hold+(blockDim.x*blockIdx.x);
    for(int stride=blockDim.x/2; stride>0; stride>>=1){
        if (tid<stride && tid+stride<size){
            blockAddress[tid]+=blockAddress[tid+stride];
        }
        __syncthreads();
    }
    if(tid==0){
        g_PartialSums[blockIdx.x]=blockAddress[0];
    }
}

__global__ void d_Commit_Dot(float* g_Partial_Sum, float* dot){
    int tid=threadIdx.x;
    float* blockAddress=g_Partial_Sum;
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
	{
		if (tid < stride)
		{
			//tid<stride ensures we do not try to access memory past the vector allocated to the block
			//tid+stride<size allows for vector sizes less than blockDim
			blockAddress[tid] += blockAddress[tid + stride];
		}
		__syncthreads();//Make all of the threads wait to go to the next iteration so the values are up to date
	}
    if(tid==0){
        *(dot)=(blockAddress[0]);
    }
}

__host__ void Dot_Helper(float* vect_1, float* vect_2, float* ref, float* dot_GPU, int size){
    float* d_v_1;
    float* d_v_2;
    float* d_hold;
    float* d_Partial;
    float* d_dot;
    int vect_size=sizeof(float)*size;
    int threads_per_block=256;
    int blocks_per_grid=size/threads_per_block+1;
    int p_sum_size=sizeof(float)*blocks_per_grid;
    HandleCUDAError(hipMalloc((void**)&d_v_1, vect_size));
    HandleCUDAError(hipMalloc((void**)&d_v_2, vect_size));
    HandleCUDAError(hipMalloc((void**)&d_hold,vect_size));
    HandleCUDAError(hipMalloc((void**)&d_Partial,p_sum_size));
    HandleCUDAError(hipMalloc((void**)&d_dot, sizeof(float)));

    HandleCUDAError(hipMemcpy(d_v_1,vect_1,vect_size,hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(d_v_2,vect_2,vect_size,hipMemcpyHostToDevice));

    d_Dot_Partial<<<blocks_per_grid,threads_per_block>>>(d_v_1,d_v_2,d_hold,d_Partial,size);
    hipDeviceSynchronize();
    d_Commit_Dot<<<1,blocks_per_grid>>>(d_Partial,d_dot);
    hipDeviceSynchronize();

    HandleCUDAError(hipMemcpy(dot_GPU,d_dot,sizeof(float),hipMemcpyDeviceToHost));
    Verify(*(dot_GPU),*(ref));

    HandleCUDAError(hipFree(d_dot));
    HandleCUDAError(hipFree(d_v_1));
    HandleCUDAError(hipFree(d_v_2));
    HandleCUDAError(hipFree(d_hold));
    HandleCUDAError(hipFree(d_Partial));
    
    hipDeviceReset();

}