#include "hip/hip_runtime.h"
#include "GPUErrors.h"
#include "Dot.h"


__global__ void d_Dot_Partial(float* in_1, float* in_2, float* hold, float* g_PartialSums, int size){
    int idx=threadIdx.x+(blockDim.x*blockIdx.x);
    int tid=threadIdx.x;
    hold[idx]=in_1[idx]*in_2[idx];
    __syncthreads();
    if (idx>=size){
        return;
    }
    float* blockAddress= hold+(blockDim.x*blockIdx.x);
    for(int stride=blockDim.x/2; stride>0; stride>>=1){
        if (tid<stride && tid+stride<size){
            blockAddress[tid]+=blockAddress[tid+stride];
        }
        __syncthreads();
    }
    if(tid==0){
        g_PartialSums[blockIdx.x]=blockAddress[0];
    }
}

__global__ void d_Commit_Dot(float* g_Partial_Sum, float* dot){
    int tid=threadIdx.x;
    float* blockAddress=g_Partial_Sum;
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
	{
		if (tid < stride)
		{
			//tid<stride ensures we do not try to access memory past the vector allocated to the block
			//tid+stride<size allows for vector sizes less than blockDim
			blockAddress[tid] += blockAddress[tid + stride];
		}
		__syncthreads();//Make all of the threads wait to go to the next iteration so the values are up to date
	}
    if(tid==0){
        *(dot)=(blockAddress[0]);
    }
}

__host__ void Dot_Helper(float* vect_1, float* vect_2, float* ref, float* dot_GPU, int size){
    float ElapsedTime{}, Elapsed_hd{}, Elapsed_dh{};
    float* d_v_1;
    float* d_v_2;
    float* d_hold;
    float* d_Partial;
    float* d_dot;
    int vect_size=sizeof(float)*size;
    int threads_per_block=256;
    int blocks_per_grid=size/threads_per_block+1;
    int p_sum_size=sizeof(float)*blocks_per_grid;
    hipEvent_t start, stop;
    hipEvent_t start_dh, stop_dh;
    hipEvent_t start_hd, stop_hd;
    HandleCUDAError(hipEventCreate(&start));
	HandleCUDAError(hipEventCreate(&stop));
    HandleCUDAError(hipEventCreate(&start_dh));
    HandleCUDAError(hipEventCreate(&stop_dh));
    HandleCUDAError(hipEventCreate(&start_hd));
    HandleCUDAError(hipEventCreate(&stop_hd));

    HandleCUDAError(hipMalloc((void**)&d_v_1, vect_size));
    HandleCUDAError(hipMalloc((void**)&d_v_2, vect_size));
    HandleCUDAError(hipMalloc((void**)&d_hold,vect_size));
    HandleCUDAError(hipMalloc((void**)&d_Partial,p_sum_size));
    HandleCUDAError(hipMalloc((void**)&d_dot, sizeof(float)));
    HandleCUDAError(hipEventRecord(start_hd,0));
    HandleCUDAError(hipMemcpy(d_v_1,vect_1,vect_size,hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(d_v_2,vect_2,vect_size,hipMemcpyHostToDevice));
    if (!HandleCUDAError(hipEventRecord(stop_hd, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop_hd))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&Elapsed_hd, start_hd, stop_hd))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    HandleCUDAError(hipEventRecord(start,0));
    d_Dot_Partial<<<blocks_per_grid,threads_per_block>>>(d_v_1,d_v_2,d_hold,d_Partial,size);
    hipDeviceSynchronize();
    d_Commit_Dot<<<1,blocks_per_grid>>>(d_Partial,d_dot);
    hipDeviceSynchronize();
    if (!HandleCUDAError(hipEventRecord(stop, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&ElapsedTime, start, stop))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    HandleCUDAError(hipEventRecord(start_dh,0));
    HandleCUDAError(hipMemcpy(dot_GPU,d_dot,sizeof(float),hipMemcpyDeviceToHost));
    if (!HandleCUDAError(hipEventRecord(stop_dh, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop_dh))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&Elapsed_dh, start_dh, stop_dh))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    float total_time=Elapsed_dh+Elapsed_hd;
    float bytes_transferred=(2*vect_size+sizeof(float))*1.0f;
    float throughput=(bytes_transferred*1e-6)/(total_time);
    cout<<"Device to Host: "<<Elapsed_dh<<" ms"<<endl;
    cout<<"Host to Device: "<<Elapsed_hd<<" ms"<<endl;
    cout<< "GPU CG Memory elapsed time: "<<total_time<< " ms"<<endl;
    cout<< "GPU CG Exec elapsed time: "<<ElapsedTime<< " ms"<<endl;
    cout<< "GPU CG total elapsed time: "<<ElapsedTime+total_time<< " ms"<<endl;
    cout<<"GPU throughput: "<<throughput<< "GB/s"<<endl;
    Verify(*(dot_GPU),*(ref));

    HandleCUDAError(hipFree(d_dot));
    HandleCUDAError(hipFree(d_v_1));
    HandleCUDAError(hipFree(d_v_2));
    HandleCUDAError(hipFree(d_hold));
    HandleCUDAError(hipFree(d_Partial));
    
    hipDeviceReset();

}