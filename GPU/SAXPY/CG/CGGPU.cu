#include "hip/hip_runtime.h"
#include "GPUErrors.h"
#include "CG.h"

__global__ void d_Dot_Partial(float* in_1, float* in_2, float* hold, float* g_PartialSums, int size){
    int idx=threadIdx.x+(blockDim.x*blockIdx.x);
    int tid=threadIdx.x;
    hold[idx]=in_1[idx]*in_2[idx];
    __syncthreads();
    if (idx>=size){
        return;
    }
    float* blockAddress= hold+(blockDim.x*blockIdx.x);
    for(int stride=blockDim.x/2; stride>0; stride>>=1){
        if (tid<stride && tid+stride<size){
            blockAddress[tid]+=blockAddress[tid+stride];
        }
        __syncthreads();
    }
    if(tid==0){
        g_PartialSums[blockIdx.x]=blockAddress[0];
    }
}

__global__ void d_Commit_Dot(float* g_Partial_Sum, float* dot, int* flag){
    int tid=threadIdx.x;
    float* blockAddress=g_Partial_Sum;
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
	{
		if (tid < stride)
		{
			//tid<stride ensures we do not try to access memory past the vector allocated to the block
			//tid+stride<size allows for vector sizes less than blockDim
			blockAddress[tid] += blockAddress[tid + stride];
		}
		__syncthreads();//Make all of the threads wait to go to the next iteration so the values are up to date
	}
    if(tid==0){
        *(dot)=(blockAddress[0]);
        if(fabsf(*dot)<1e-8){
            *flag=0;
        }
    }
}

__global__ void VectAdd(float* g_A, float* g_B, float* g_C, int size){
    int idx=threadIdx.x+(blockDim.x*blockIdx.x);
    if(idx<size){
        g_C[idx]=g_A[idx]+g_B[idx];
    }
}

__global__ void Const_Vect_Mult(float* vect, float* out, float* scalar, int size){
    int idx =threadIdx.x+(blockDim.x*blockIdx.x);
    if(idx<size){
        out[idx]=*scalar*vect[idx];
    }
}


__global__ void MatrixVectorMult(float* g_Matrix, float* g_V, float* g_P, const int Size) {
	int row = threadIdx.x + (blockDim.x * blockIdx.x);//We are providing this automatic variable to allow each thread to identify its location
	//Each thread will calculate each entry in our resulting vector
	//To do so, each thread will extract a row of g_Matrix to do with the vector g_V
	float fSum = 0.0f;//We create an automatic variable fSum for each thread to lower memory accesses in the for loop
	//We are going to use fSum instead of writing g_P[row]+=....
	if (row < Size) {
		//We are trying to ensure we are not using more threads than data we have
		for (int k{}; k < Size;k++) {
			fSum += g_Matrix[row * Size + k] * g_V[k];//Here we are dotting the row of g_matrix(corresponding to the index of each thread) with g_V
		}
		g_P[row] = fSum;//We now assign the row_th entry of g_P the value fSum, i.e., our dot product
	}
}

__global__ void comp_lamba(float* in, float* in_2, float* out,int size, int flag){
    int idx =threadIdx.x+(blockDim.x*blockIdx.x);
    if(idx<size){
        if(flag==1){
            out[idx]=in[idx]*in_2[idx];
        }
        else{
            out[idx]=-in[idx]*in_2[idx];
        }
    }

}


__host__ void CG_Helper(float* A, float* ref, float* r, float* r_old, float* d, float* d_old, float* x, float* x_old, float beta, float lamdba, int size){
    float* d_Ad;
    float* lamd_d;
    float* beta_d;
    float* lambd_AD;
    float* d_A;
    float* d_r;
    float* d_r_old;
    float* d_d;
    float* d_d_old;
    float* d_x;
    float* d_x_old;
    float* d_beta;
    float* d_lambda;
    float* d_neg_lambda;
    float* temp_1;
    float* temp_2;
    float* d_dot_partial_1;
    float* d_dot_partial_2;
    float* d_hold_1;
    float* d_hold_2;
    int* flag;
    int* flag_2;
    int threads_per_block=128;
    int blocks_per_grid=size/threads_per_block+1;
    int mat_size=size*size*sizeof(float);
    int vect_size=size*sizeof(float);
    int var_size=sizeof(float);
    int flag_size=sizeof(int);
    int p_sum_size=sizeof(float)*blocks_per_grid;
    int host_flag=1;
    
    HandleCUDAError(hipMalloc((void**) &d_A,mat_size));
    HandleCUDAError(hipMalloc((void**) &d_r,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_r_old,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_d,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_d_old,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_x,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_x_old,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_beta,var_size));
    HandleCUDAError(hipMalloc((void**) &d_lambda,var_size));
    HandleCUDAError(hipMalloc((void**) &d_neg_lambda,var_size));
    HandleCUDAError(hipMalloc((void**) &temp_1,var_size));
    HandleCUDAError(hipMalloc((void**) &temp_2,var_size));
    HandleCUDAError(hipMalloc((void**) &flag,flag_size));
    HandleCUDAError(hipMalloc((void**) &flag_2,flag_size));
    HandleCUDAError(hipMalloc((void**) &d_Ad,vect_size));
    HandleCUDAError(hipMalloc((void**) &lamd_d,vect_size));
    HandleCUDAError(hipMalloc((void**) &beta_d,mat_size));
    HandleCUDAError(hipMalloc((void**) &lambd_AD,mat_size));
    HandleCUDAError(hipMalloc((void**) &d_dot_partial_1,p_sum_size));
    HandleCUDAError(hipMalloc((void**) &d_dot_partial_2,p_sum_size));
    HandleCUDAError(hipMalloc((void**) &d_hold_1,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_hold_2,vect_size));
    HandleCUDAError(hipMemcpy(d_A,A,mat_size,hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(d_r_old,r_old,vect_size,hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(d_d_old,d_old,vect_size,hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(d_x_old,x_old,vect_size,hipMemcpyHostToDevice));

    hipStream_t dot_1;
    hipStream_t dot_2;

    hipStreamCreate(&dot_1);
    hipStreamCreate(&dot_2);
    while(host_flag){
        MatrixVectorMult<<<blocks_per_grid,threads_per_block>>>(d_A,d_d_old,d_Ad,size);
        hipDeviceSynchronize();
        d_Dot_Partial<<<blocks_per_grid,threads_per_block,0,dot_1>>>(d_r_old,d_r_old,d_hold_1,d_dot_partial_1,size);
        d_Dot_Partial<<<blocks_per_grid,threads_per_block,0,dot_2>>>(d_r_old,d_r_old,d_hold_1,d_dot_partial_1,size);
        hipStreamSynchronize(dot_1);
        hipStreamSynchronize(dot_2);
        d_Commit_Dot<<<1,blocks_per_grid,0,dot_1>>>(d_dot_partial_1,temp_1,flag);
        d_Commit_Dot<<<1,blocks_per_grid,0,dot_2>>>(d_dot_partial_2,temp_2,flag_2);
        hipStreamSynchronize(dot_1);
        hipStreamSynchronize(dot_2);
        comp_lamba<<<1,1,0,dot_1>>>(temp_1,temp_2,d_lambda,1,1);
        comp_lamba<<<1,1,0,dot_1>>>(temp_1,temp_2,d_lambda,1,0);
        hipStreamSynchronize(dot_1);
        hipStreamSynchronize(dot_2);
    }


}