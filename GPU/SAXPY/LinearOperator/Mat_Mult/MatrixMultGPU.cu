#include "hip/hip_runtime.h"
﻿#include "MatrixMult.h"
#include "GPUErrors.h"
__global__ void NaiveMult(float* g_A, float* g_B, float* g_C, const int ny, const int nx) {
	int row = threadIdx.y + (blockIdx.y * blockDim.y);
	int col = threadIdx.x + (blockIdx.x * blockDim.x);
	float fSum = 0.0f;
	//This conditional is for debugging, even though done on the device

	if (row < ny && col < nx) { 
		for (int k = 0; k < nx; k++) {
			int idx_A = abs(row-k);
			int idx_B = abs(k-col);
			fSum += g_A[idx_A] * g_B[idx_B];
		}
		g_C[row * (nx) + col] = fSum;
	}
}



__host__ void gpuMultHelper(float* h_A, float* h_B, float* h_C, float* ref, const int ny, const int nx)
{
	float* d_A, * d_B, * d_C;
	const int MatrixSizeInBytes = nx * sizeof(float);//rowsxcolsxnumber of bytes
	const int C_size_in_Bytes=sizeof(float)*(nx*(ny));
	//Allocate device memory on the global memory
	cout<<(nx*(nx+1))/2<<endl;
	
	HandleCUDAError(hipMalloc((void**)&d_A, MatrixSizeInBytes));
	HandleCUDAError(hipMalloc((void**)&d_B, MatrixSizeInBytes));
	HandleCUDAError(hipMalloc((void**)&d_C, C_size_in_Bytes));


	//transfer data from CPU Memory to GPU Memory
	chrono::time_point<std::chrono::system_clock> start, end;
	start = std::chrono::system_clock::now();
	if(!HandleCUDAError(hipMemcpy(d_A, h_A, MatrixSizeInBytes, hipMemcpyHostToDevice))){
		cout<<"cannot transfer A"<<endl;
	}

	if(!HandleCUDAError(hipMemcpy(d_B, h_B, MatrixSizeInBytes, hipMemcpyHostToDevice))){
		cout<<"cannot transfer B"<<endl;
	}
	end = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	cout << "GPU Memory Transfer time (H to D): " << (elapsed_seconds.count() * 1000.0f) << " msecs" << endl;

	//Kernel Invoke Parameters - 2D Grid and 2D Blocks
	int dimx = 16;
	int dimy = 16;
	dim3 block(dimx, dimy);//16x16 block
	//Never use the maximum number of threads in a block, the standard value is 256 or less
	dim3 grid((ny+block.y-1)/block.y, (nx + block.x - 1) / block.x);//First says the number of blocks in the y direction, this ensures number of blocks can handle data
	//We do not need to cast since the structure is with ints
	cout << "\t2D Grid Dimension" << endl;
	cout << "\tNumber of Blocks along X dimension: " << grid.x << endl;
	cout << "\tNumber of Blocks along Y dimension: " << grid.y << endl;
	cout << "\t2D Block Dimension" << endl;
	cout << "\tNumber of threads along X dimension: " << block.x << endl;
	cout << "\tNumber of threads along Y dimension: " << block.y << endl;

	//Launch Multiplication kernel
	start = std::chrono::system_clock::now();
	NaiveMult << <grid, block >> > (d_A,d_B,d_C,ny,nx);
	hipDeviceSynchronize();//Recall this makes the CPU wait
	end = std::chrono::system_clock::now();
	elapsed_seconds = end - start;
	cout << "Naive Multiplication: GPU Execution time: " << (elapsed_seconds.count() * 1000.0f) << " msecs" << endl;

	start = std::chrono::system_clock::now();
	//Copy product matrix to host
	if(!HandleCUDAError(hipMemcpy(h_C, d_C, C_size_in_Bytes, hipMemcpyDeviceToHost))){
		cout<<"Unable to transfer C"<<endl;
	}
	end = std::chrono::system_clock::now();
	elapsed_seconds = end - start;
	cout << "GPU Memory Transfer time (D to H): " << (elapsed_seconds.count() * 1000.0f) << " msecs" << endl;
	
	//Verification code
	Verify(h_C,ref, ny);
	
	//Release Memory and reset device
	HandleCUDAError(hipFree(d_A));
	HandleCUDAError(hipFree(d_B));
	HandleCUDAError(hipFree(d_C));

	hipDeviceReset();
}