#include "hip/hip_runtime.h"
#include "GPUErrors.h"
#include "CG.h"

__global__ void d_Dot_Partial(float* in_1, float* in_2, float* hold, float* g_PartialSums, int size){
    int idx=threadIdx.x+(blockDim.x*blockIdx.x);
    int tid=threadIdx.x;
    hold[idx]=in_1[idx]*in_2[idx];
    __syncthreads();
    if (idx>=size){
        return;
    }
    float* blockAddress= hold+(blockDim.x*blockIdx.x);
    for(int stride=blockDim.x/2; stride>0; stride>>=1){
        if (tid<stride && tid+stride<size){
            blockAddress[tid]+=blockAddress[tid+stride];
        }
        __syncthreads();
    }
    if(tid==0){
        g_PartialSums[blockIdx.x]=blockAddress[0];
    }
}

__global__ void d_Commit_Dot(float* g_Partial_Sum, float* dot){
    int tid=threadIdx.x;
    float* blockAddress=g_Partial_Sum;
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
	{
		if (tid < stride)
		{
			//tid<stride ensures we do not try to access memory past the vector allocated to the block
			//tid+stride<size allows for vector sizes less than blockDim
			blockAddress[tid] += blockAddress[tid + stride];
		}
		__syncthreads();//Make all of the threads wait to go to the next iteration so the values are up to date
	}
    if(tid==0){
        *(dot)=(blockAddress[0]);
    }
}

__global__ void VectAdd(float* g_A, float* g_B, float* g_C, int size){
    int idx=threadIdx.x+(blockDim.x*blockIdx.x);
    if(idx<size){
        g_C[idx]=g_A[idx]+g_B[idx];
    }
}

__global__ void d_Const_Vect_Mult(float* vect, float* out, float* scalar, int size){
    int idx =threadIdx.x+(blockDim.x*blockIdx.x);
    if(idx<size){
        out[idx]=(*scalar)*vect[idx];
    }
}


__global__ void MatrixVectorMult(float* g_Matrix, float* g_V, float* g_P, const int Size) {
	int row = threadIdx.x + (blockDim.x * blockIdx.x);//We are providing this automatic variable to allow each thread to identify its location
	//Each thread will calculate each entry in our resulting vector
	//To do so, each thread will extract a row of g_Matrix to do with the vector g_V
	float fSum = 0.0f;//We create an automatic variable fSum for each thread to lower memory accesses in the for loop
	//We are going to use fSum instead of writing g_P[row]+=....
	if (row < Size) {
		//We are trying to ensure we are not using more threads than data we have
		for (int k{}; k < Size;k++) {
            int idx=abs(row-k);
			fSum += g_Matrix[idx] * g_V[k];//Here we are dotting the row of g_matrix(corresponding to the index of each thread) with g_V
		}
		g_P[row] = fSum;//We now assign the row_th entry of g_P the value fSum, i.e., our dot product
	}
}

__global__ void comp_lamba(float* in, float* in_2, float* out,int size, int flag){
    int idx =threadIdx.x+(blockDim.x*blockIdx.x);
    if(idx<size){
        if(flag==1){
            *out=*in/(*in_2);
        }
        else{
            *out=-*in/(*in_2);
        }
    }

}

__global__ void Copy(float* in, float* out, int size){
    int idx=threadIdx.x+(blockIdx.x*blockDim.x);
    if (idx<size){
        out[idx]=in[idx];
    }
}


__host__ void CG_Helper(float* A, float* ref, float* r, float* r_old, float* d, float* d_old, float* x, float* x_old, float beta, float lamdba, int size, int iter){
    float ElapsedTime{}, Elapsed_hd{}, Elapsed_dh{};
    float* d_Ad;
    float* lamd_d;
    float* beta_d;
    float* lambd_AD;
    float* d_A;
    float* d_r;
    float* d_r_old;
    float* d_d;
    float* d_d_old;
    float* d_x;
    float* d_x_old;
    float* d_beta;
    float* d_lambda;
    float* d_neg_lambda;
    float* temp_1;
    float* temp_2;
    float* d_dot_partial_1;
    float* d_dot_partial_2;
    float* d_hold_1;
    float* d_hold_2;
    int threads_per_block=128;
    int blocks_per_grid=(size/threads_per_block)+1;
    int vect_size=size*sizeof(float);
    int var_size=sizeof(float);
    int p_sum_size=sizeof(float)*blocks_per_grid;
    HandleCUDAError(hipMalloc((void**) &d_A,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_r,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_r_old,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_d,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_d_old,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_x,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_x_old,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_beta,var_size));
    HandleCUDAError(hipMalloc((void**) &d_lambda,var_size));
    HandleCUDAError(hipMalloc((void**) &d_neg_lambda,var_size));
    HandleCUDAError(hipMalloc((void**) &temp_1,var_size));
    HandleCUDAError(hipMalloc((void**) &temp_2,var_size));
    HandleCUDAError(hipMalloc((void**) &d_Ad,vect_size));
    HandleCUDAError(hipMalloc((void**) &lamd_d,vect_size));
    HandleCUDAError(hipMalloc((void**) &beta_d,vect_size));
    HandleCUDAError(hipMalloc((void**) &lambd_AD,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_dot_partial_1,p_sum_size));
    HandleCUDAError(hipMalloc((void**) &d_dot_partial_2,p_sum_size));
    HandleCUDAError(hipMalloc((void**) &d_hold_1,vect_size));
    HandleCUDAError(hipMalloc((void**) &d_hold_2,vect_size));

    hipStream_t dot_1;
    hipStream_t dot_2;
    hipStream_t copy_1;
    hipStream_t copy_2;
    hipStream_t copy_3;
    hipStreamCreate(&dot_1);
    hipStreamCreate(&dot_2);
    hipStreamCreate(&copy_1);
    hipStreamCreate(&copy_2);
    hipStreamCreate(&copy_3);
    hipEvent_t start, stop;
    hipEvent_t start_dh, stop_dh;
    hipEvent_t start_hd, stop_hd;
	HandleCUDAError(hipEventCreate(&start));
	HandleCUDAError(hipEventCreate(&stop));
    HandleCUDAError(hipEventCreate(&start_dh));
    HandleCUDAError(hipEventCreate(&stop_dh));
    HandleCUDAError(hipEventCreate(&start_hd));
    HandleCUDAError(hipEventCreate(&stop_hd));
    HandleCUDAError(hipEventRecord(start_hd,0));
    HandleCUDAError(hipMemcpy(d_A,A,vect_size,hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(d_r_old,r_old,vect_size,hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(d_d_old,d_old,vect_size,hipMemcpyHostToDevice));
    HandleCUDAError(hipMemcpy(d_x_old,x_old,vect_size,hipMemcpyHostToDevice));
    if (!HandleCUDAError(hipEventRecord(stop_hd, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop_hd))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&Elapsed_hd, start_hd, stop_hd))) {
		cout << "Unable to find elapsed time between events" << endl;
	}

    HandleCUDAError(hipEventRecord(start,0));
    for(int i{}; i<iter; i++){
        MatrixVectorMult<<<blocks_per_grid,threads_per_block>>>(d_A,d_d_old,d_Ad,size);
        hipDeviceSynchronize();
        d_Dot_Partial<<<blocks_per_grid,threads_per_block,0,dot_1>>>(d_r_old,d_r_old,d_hold_1,d_dot_partial_1,size);
        d_Dot_Partial<<<blocks_per_grid,threads_per_block,0,dot_2>>>(d_d_old,d_Ad,d_hold_2,d_dot_partial_2,size);
        hipStreamSynchronize(dot_1);
        hipStreamSynchronize(dot_2);
        d_Commit_Dot<<<1,blocks_per_grid,0,dot_1>>>(d_dot_partial_1,temp_1);
        d_Commit_Dot<<<1,blocks_per_grid,0,dot_2>>>(d_dot_partial_2,temp_2);
        hipStreamSynchronize(dot_1);
        hipStreamSynchronize(dot_2);

        comp_lamba<<<1,1,0,dot_1>>>(temp_1,temp_2,d_lambda,1,1);
        comp_lamba<<<1,1,0,dot_2>>>(temp_1,temp_2,d_neg_lambda,1,0);
        hipStreamSynchronize(dot_1);
        hipStreamSynchronize(dot_2);
        //Need to do scalar mult
        d_Const_Vect_Mult<<<blocks_per_grid,threads_per_block,0,dot_1>>>(d_d_old,lamd_d,d_lambda,size);
        d_Const_Vect_Mult<<<blocks_per_grid,threads_per_block,0,dot_2>>>(d_Ad,lambd_AD,d_neg_lambda,size);
        hipStreamSynchronize(dot_1);
        hipStreamSynchronize(dot_2);
        VectAdd<<<blocks_per_grid,threads_per_block,0,dot_1>>>(d_x_old,lamd_d,d_x,size);
        VectAdd<<<blocks_per_grid,threads_per_block,0,dot_2>>>(d_r_old,lambd_AD,d_r,size);
        hipStreamSynchronize(dot_1);
        hipStreamSynchronize(dot_2);

        d_Dot_Partial<<<blocks_per_grid,threads_per_block>>>(d_r,d_r,d_hold_2,d_dot_partial_2,size);
        hipDeviceSynchronize();

        d_Commit_Dot<<<1,blocks_per_grid>>>(d_dot_partial_2,temp_2);
        hipDeviceSynchronize();

        comp_lamba<<<1,1>>>(temp_2,temp_1,d_beta,1,1);
        hipDeviceSynchronize();

        d_Const_Vect_Mult<<<blocks_per_grid,threads_per_block>>>(d_d_old,beta_d,d_beta,size);
        hipDeviceSynchronize();

        VectAdd<<<blocks_per_grid,threads_per_block>>>(d_r,beta_d,d_d,size);
        hipDeviceSynchronize();

        Copy<<<blocks_per_grid,threads_per_block,0,copy_1>>>(d_d,d_d_old,size);
        Copy<<<blocks_per_grid,threads_per_block,0,copy_2>>>(d_r,d_r_old,size);
        Copy<<<blocks_per_grid,threads_per_block,0,copy_3>>>(d_x,d_x_old,size);
        hipStreamSynchronize(copy_1);
        hipStreamSynchronize(copy_2);
        hipStreamSynchronize(copy_3);

    }
    hipStreamDestroy(dot_1);
    hipStreamDestroy(dot_2);
    hipStreamDestroy(copy_1);
    hipStreamDestroy(copy_2);
    hipStreamDestroy(copy_3);
    if (!HandleCUDAError(hipEventRecord(stop, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&ElapsedTime, start, stop))) {
		cout << "Unable to find elapsed time between events" << endl;
	}
    HandleCUDAError(hipEventRecord(start_dh,0));
    HandleCUDAError(hipMemcpy(x,d_x,vect_size,hipMemcpyDeviceToHost));
    if (!HandleCUDAError(hipEventRecord(stop_dh, 0))) {
		cout << "Unable to perform event records for stop" << endl;
	}
	//Synchronize the stop event
	if (!HandleCUDAError(hipEventSynchronize(stop_dh))) {
		cout << "Unable to perform stream synch with stream_4" << endl;
	}
	//Save the elapsed time
	if (!HandleCUDAError(hipEventElapsedTime(&Elapsed_dh, start_dh, stop_dh))) {
		cout << "Unable to find elapsed time between events" << endl;
	}

    float total_time=Elapsed_dh+Elapsed_hd;
    float bytes_transferred=(5*vect_size)*1.0f;
    float throughput=(bytes_transferred*1e-6)/(total_time);
    cout<< "GPU CG Memory elapsed time: "<<total_time<< " ms"<<endl;
    cout<< "GPU CG Exec elapsed time: "<<ElapsedTime<< " ms"<<endl;
    cout<< "GPU CG total elapsed time: "<<ElapsedTime+total_time<< " ms"<<endl;
    cout<<"GPU throughput: "<<throughput<< "GB/s"<<endl;

    Verify(x,ref,size);
    HandleCUDAError(hipFree(d_A));
    HandleCUDAError(hipFree(d_r));
    HandleCUDAError(hipFree(d_r_old));
    HandleCUDAError(hipFree(d_d));
    HandleCUDAError(hipFree(d_d_old));
    HandleCUDAError(hipFree(d_x));
    HandleCUDAError(hipFree(d_x_old));
    HandleCUDAError(hipFree(d_beta));
    HandleCUDAError(hipFree(d_neg_lambda));
    HandleCUDAError(hipFree(d_lambda));
    HandleCUDAError(hipFree(temp_1));
    HandleCUDAError(hipFree(temp_2));
    HandleCUDAError(hipFree(d_hold_1));
    HandleCUDAError(hipFree(d_hold_2));
    HandleCUDAError(hipFree(d_Ad));
    HandleCUDAError(hipFree(d_dot_partial_1));
    HandleCUDAError(hipFree(d_dot_partial_2));
    HandleCUDAError(hipFree(lambd_AD));
    HandleCUDAError(hipFree(lamd_d));
    HandleCUDAError(hipFree(beta_d));
    hipDeviceReset();


}